// Redistribution and use in source and binary forms, with or without modification, are permitted
// provided that the following conditions are met:
//     * Redistributions of source code must retain the above copyright notice, this list of
//       conditions and the following disclaimer.
//     * Redistributions in binary form must reproduce the above copyright notice, this list of
//       conditions and the following disclaimer in the documentation and/or other materials
//       provided with the distribution.
//     * Neither the name of the NVIDIA CORPORATION nor the names of its contributors may be used
//       to endorse or promote products derived from this software without specific prior written
//       permission.
// 
// THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR
// IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND
// FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE
// FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
// BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS;
// OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT,
// STRICT LIABILITY, OR TOR (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
// OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.


#include <hip/hip_runtime.h>
#include <iostream>
#include <map>
#include <stdexcept>
#include <string>

#include <nvml.h>

const int max_slices = 7;

std::map<unsigned int, unsigned int> id_slice = {
    { 0, 4 },
    { 5, 3 },
    { 9, 2 },
    { 14, 1 },
    { 19, 0 }
};

template<typename T>
void check( T const &err_code, std::string const &file, int const &line ) {
    if ( err_code ) {
        hipDeviceReset( );
        std::string str = nvmlErrorString( err_code );
        throw std::runtime_error( str + " in " + file + " at line " +
                                  std::to_string( line ) );
    }
}

#define checkNVMLErrors( errCode ) check( errCode, __FILE__, __LINE__ )

int main( int argc, char **argv ) {

	unsigned int mig_mode { static_cast<unsigned int>(std::stoi(argv[1])) };
	unsigned int count { static_cast<unsigned int>(std::stoi(argv[2])) };

	// Initialize NVML library
	checkNVMLErrors( nvmlInit( ) );

	// Query device handle
	nvmlDevice_t       device_ {};
	checkNVMLErrors( nvmlDeviceGetHandleByIndex( 0, &device_ ) );

	// Query device name
	char name[NVML_DEVICE_NAME_BUFFER_SIZE];
	checkNVMLErrors( nvmlDeviceGetName( device_, name, NVML_DEVICE_NAME_BUFFER_SIZE ) );

	// Query compute capability
	int major {};
	int minor {};
	checkNVMLErrors( nvmlDeviceGetCudaComputeCapability( device_, &major, &minor ) );

	// Query MIG mode
	unsigned int currentMode {};
	unsigned int pendingMode {};
	checkNVMLErrors( nvmlDeviceGetMigMode( device_, &currentMode, &pendingMode ) );

	// Query GPU Instances used
	nvmlGpuInstanceProfileInfo_t gid_profile {};
	checkNVMLErrors( nvmlDeviceGetGpuInstanceProfileInfo( device_, id_slice.at(mig_mode), &gid_profile ) );

	// Query GPU Instances used
	nvmlGpuInstance_t gids[max_slices];
	unsigned int g_used {};
	checkNVMLErrors( nvmlDeviceGetGpuInstances( device_, gid_profile.id, gids, &g_used ) );

	// Query GPU Instance Info
	nvmlGpuInstanceInfo_t gid_info {};
	checkNVMLErrors( nvmlGpuInstanceGetInfo( gids[count], &gid_info ) );

	// Query remaining capacity
	unsigned int g_avail {};
	checkNVMLErrors( nvmlDeviceGetGpuInstanceRemainingCapacity( device_, gid_profile.id, &g_avail ) );

	// Get Compute Instance Info
	nvmlComputeInstanceProfileInfo_t cid_profile {};
	checkNVMLErrors( nvmlGpuInstanceGetComputeInstanceProfileInfo( gids[count], id_slice.at(mig_mode), 0, &cid_profile) );

	// Query Compute Instances used
	nvmlComputeInstance_t cids {};
	unsigned int c_used {};
	checkNVMLErrors( nvmlGpuInstanceGetComputeInstances( gids[count], cid_profile.id, &cids, &c_used ) );

	// Query Compute Info
	nvmlComputeInstanceInfo_t cid_info {};
	checkNVMLErrors( nvmlComputeInstanceGetInfo( cids, &cid_info ) );

	// Query remaining capacity
	unsigned int c_avail {};
	checkNVMLErrors( nvmlGpuInstanceGetComputeInstanceRemainingCapacity( gids[count], 0, &c_avail ) );

	printf("\n");
	printf("Name: %s\n", name);
	printf("Compute Capability: %d%d\n", major, minor);
	printf("MIG Enabled: %s\n", (currentMode ? "True" : "False"));
	printf("GPU Profile Id: %d\n", gid_info.profileId);
	printf("GPU Instance Id: %d\n", gid_info.id);
	printf("GPU Instances Used: %d\n", g_used);
	printf("GPU Instances Avail: %d\n", g_avail);
	printf("Compute Profile Id: %d\n", cid_profile.id);
	printf("Compute Instance Id: %d\n", cid_info.id);
	printf("Compute Instances Used: %d\n", g_used);
	printf("Compute Instances Avail: %d\n", g_avail);
	printf("\n");
	
	// Shutdown NVML
	checkNVMLErrors( nvmlShutdown( ) );

	return ( EXIT_SUCCESS );
}
